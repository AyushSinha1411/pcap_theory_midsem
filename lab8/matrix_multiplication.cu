
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 4  // Matrix dimension (N x N)

// Kernel for row-wise matrix multiplication (one thread per row)
__global__ void matrixMulRow(int *a, int *b, int *c, int n) {
    int row = threadIdx.x;  // Thread ID represents the row
    if (row < n) {
        for (int col = 0; col < n; col++) {
            int sum = 0;
            for (int k = 0; k < n; k++) {
                sum += a[row * n + k] * b[k * n + col];
            }
            c[row * n + col] = sum;
        }
    }
}

// Kernel for column-wise matrix multiplication (one thread per column)
__global__ void matrixMulCol(int *a, int *b, int *c, int n) {
    int col = threadIdx.x;  // Thread ID represents the column
    if (col < n) {
        for (int row = 0; row < n; row++) {
            int sum = 0;
            for (int k = 0; k < n; k++) {
                sum += a[row * n + k] * b[k * n + col];
            }
            c[row * n + col] = sum;
        }
    }
}

// Kernel for element-wise matrix multiplication (one thread per element)
__global__ void matrixMulElement(int *a, int *b, int *c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < n && col < n) {
        int sum = 0;
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

// Function to print matrix
void printMatrix(int *matrix, int n) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%d\t", matrix[i * n + j]);
        }
        printf("\n");
    }
}

int main() {
    int size = N * N * sizeof(int);
    int *h_a, *h_b, *h_c;  // Host matrices
    int *d_a, *d_b, *d_c;  // Device matrices
    
    // Allocate host memory
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);
    
    // Initialize matrices with sample data
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_a[i * N + j] = i + j + 1;  // Simple pattern for matrix A
            h_b[i * N + j] = i + j + 2;  // Simple pattern for matrix B
        }
    }
    
    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    printf("\nMatrix A:\n");
    printMatrix(h_a, N);
    printf("\nMatrix B:\n");
    printMatrix(h_b, N);
    
    // 1. Row-wise multiplication
    printf("\n1. Row-wise Multiplication (one thread per row):\n");
    matrixMulRow<<<1, N>>>(d_a, d_b, d_c, N);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    printMatrix(h_c, N);
    
    // 2. Column-wise multiplication
    printf("\n2. Column-wise Multiplication (one thread per column):\n");
    matrixMulCol<<<1, N>>>(d_a, d_b, d_c, N);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    printMatrix(h_c, N);
    
    // 3. Element-wise multiplication
    printf("\n3. Element-wise Multiplication (one thread per element):\n");
    dim3 threadsPerBlock(2, 2);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matrixMulElement<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, N);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    printMatrix(h_c, N);
    
    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    
    return 0;
}
