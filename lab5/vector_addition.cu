#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addVectorsKernelBlockSizeN(float *a, float *b, float *c, int n) {
    int idx = threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

__global__ void addVectorsKernelNThreads(float *a, float *b, float *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    int n = 1024; // Length of the vectors
    size_t size = n * sizeof(float);

    // Allocate memory on the host
    float *h_a = (float *)malloc(size);
    float *h_b = (float *)malloc(size);
    float *h_c = (float *)malloc(size);

    // Initialize vectors
    for (int i = 0; i < n; i++) {
        h_a[i] = (float)i;
        h_b[i] = (float)i;
    }

    // Allocate memory on the device
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy vectors from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch kernel with block size as n
    addVectorsKernelBlockSizeN<<<1, n>>>(d_a, d_b, d_c, n);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print results for block size as n
    printf("Results for block size as n:\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_c[i]);
    }
    printf("\n");

    // Launch kernel with n threads
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    addVectorsKernelNThreads<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print results for n threads
    printf("Results for n threads:\n");
    for (int i = 0; i < 10; i++) {
        printf("%f ", h_c[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}
